#include "hip/hip_runtime.h"
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>
#include <cstddef>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32

template <typename T>
__forceinline__ __device__ T add_eps(T x) {
  return fabsf(x) > LN_EPSILON ? x : (x < 0 ? -LN_EPSILON : LN_EPSILON);
}

/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size * seq_len, hidden_size], ln result.
vars: [batch_size * seq_len], variance per token
means: [batch_size * seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute x and x^2 with reinterpret_cast by casting to float4 for speedup
  // 2. Compute reduce sum with blockReduce and add epsilon with LN_EPSILON
  // 3. Compute layernorm result with reinterpret_cast by casting to float4 for speedup
  
  // Step 1
  // if (isnan(inp[0]) || isnan(inp[1])) {
  //   printf("NaN Detected! \n");
  // }

  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;

  // if (isnan(inp_f4[0].x) || isnan(inp_f4[0].y) || isnan(inp_f4[0].z) || isnan(inp_f4[0].w)) {
  //   printf("NaN Detected! inp_f4 \n");
  //   printf("inp: %f, %f, %f, %f\n", inp[0], inp[1], inp[2], inp[3]);
  // }

  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];

    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum += val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;

    // if (isnan(l_sum) || isnan(l_square_sum)) {
    //   printf("val.x: %f, val.y: %f, val.z: %f, val.w: %f\n", val.x, val.y, val.z, val.w);
    // }
  }

  // Step 2
  float mean_dim = float(hidden_size) * 4.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);

  // if (isnan(reduce_val[0]) || isnan(reduce_val[1])) {
  //   printf("------------------------------------------------------- \n");
  //   printf("val.x: %f, val.y: %f, val.z: %f, val.w: %f\n", val.x, val.y, val.z, val.w);
  //   printf("l_sum: %f, l_square_sum: %f\n", l_sum, l_square_sum);
  //   printf("reduce_val[0]: %f, reduce_val[1]: %f\n", reduce_val[0], reduce_val[1]);
  //   printf("mean_dim: %f\n", mean_dim);
  //   // Optionally, print more diagnostics
  // }

  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    // if (isnan(s_mean) || isnan(s_var)) {
    //     printf("NaN Detected! \n");
    //     printf("Block: %d, Thread: %d\n", blockIdx.x, threadIdx.x);
    //     printf("l_sum: %f, l_square_sum: %f\n", l_sum, l_square_sum);
    //     printf("reduce_val[0]: %f, reduce_val[1]: %f\n", reduce_val[0], reduce_val[1]);
    //     printf("mean_dim: %f\n", mean_dim);
    //     printf("s_mean: %f, s_var: %f\n", s_mean, s_var);
    //     // Optionally, print more diagnostics
    // }
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // Step 3
  float4 *output_f4 = reinterpret_cast<float4 *>(ln_res) + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 vscale = __ldg(reinterpret_cast<const float4 *>(scale) + idx);
    float4 vbias = __ldg(reinterpret_cast<const float4 *>(bias) + idx);
    float4 val = inp_f4[idx];
    val.x = (val.x - s_mean) * s_var * vscale.x + vbias.x;
    val.y = (val.y - s_mean) * s_var * vscale.y + vbias.y;
    val.z = (val.z - s_mean) * s_var * vscale.z + vbias.z;
    val.w = (val.w - s_mean) * s_var * vscale.w + vbias.w;
    output_f4[idx] = val;
  }
  
  /// END ASSIGN3_2
}

extern "C" {
void launch_layernorm(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  int float_size = sizeof(float);
  int input_size = batch_size * hidden_dim * float_size;
  int scale_size = hidden_dim * float_size;
  int bias_size = hidden_dim * float_size;
  int output_size = batch_size * hidden_dim * float_size;
  int mean_size = batch_size * float_size;
  int var_size = batch_size * float_size;


  float *d_ln_res, *d_vars, *d_means, *d_inp, *d_scale, *d_bias;
  hipMalloc((void **)&d_ln_res, output_size);
  hipMalloc((void **)&d_vars, var_size);
  hipMalloc((void **)&d_means, mean_size);
  hipMalloc((void **)&d_inp, input_size);
  hipMalloc((void **)&d_scale, scale_size);
  hipMalloc((void **)&d_bias, bias_size);

  hipMemcpy(d_inp, inp, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_scale, scale, scale_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, bias_size, hipMemcpyHostToDevice);

  // For using float4
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      d_ln_res, d_vars, d_means, d_inp, d_scale, d_bias, hidden_dim);

  // Copy back to the host
  hipMemcpy(ln_res, d_ln_res, output_size, hipMemcpyDeviceToHost);
  hipMemcpy(vars, d_vars, var_size, hipMemcpyDeviceToHost);
  hipMemcpy(means, d_means, mean_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm Error: %s\n", hipGetErrorString(err));
    // Handle the error (e.g., by exiting the program)
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_ln_res);
  hipFree(d_vars);
  hipFree(d_means);
  hipFree(d_inp);
  hipFree(d_scale);
  hipFree(d_bias);

}
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma

@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad,
                                        const T *inp, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, int rows, int width) {

  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute the partial gradients by looping across inp rows
  // 2. Store the partial gradients in the shared memory arrays
  // 3. Compute the reduce sum of the shared memory arrays with g.shfl_down
  // 4. Assign the final result to the correct position in the global output

  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = threadIdx.y * width + idx;
  int y_stride = width * TILE_DIM;

  // Step 1
  float dbetta = 0;
  float dgamma = 0;
  float dout, val;

  float thread_cmax_g = 0;
  if (idx < width) {
    if (means == nullptr) {
      float vbetta = (float)betta[idx];
      float vgamma = (float)gamma[idx];
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];

        val = (float)inp[offset];
        dbetta += dout;
        dgamma += ((val - vbetta) / add_eps(vgamma) * dout);
        offset += y_stride;
      }
    } 
    else {
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];
        val = (float)inp[offset];
        dbetta += dout;
        dgamma += ((val - (float)means[r]) * rsqrtf((float)vars[r] + LN_EPSILON) * dout);
        offset += y_stride;
      }
    }
  }

  // Step 2
  __shared__ float block_cmax_g;
  if (threadIdx.x == 0 && threadIdx.y == 0) block_cmax_g = 0;

  betta_buffer[threadIdx.x][threadIdx.y] = dbetta;
  gamma_buffer[threadIdx.x][threadIdx.y] = dgamma;
  __syncthreads();

  if (thread_cmax_g != 0) {
    atomicAdd(&block_cmax_g, thread_cmax_g);
  }

  float s1 = betta_buffer[threadIdx.y][threadIdx.x];
  float s2 = gamma_buffer[threadIdx.y][threadIdx.x];
  __syncthreads();

  // Step 3
  for (int i = 1; i < TILE_DIM; i <<= 1) {
    s1 += g.shfl_down(s1, i);
    s2 += g.shfl_down(s2, i);
  }

  // Step 4
  int pos = blockIdx.x * TILE_DIM + threadIdx.y;
  if (threadIdx.x == 0 && idx < width) {
    betta_grad[pos] = s1;
    gamma_grad[pos] = s2;
  }
  /// END ASSIGN3_2
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad, const T *inp,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute dxhat=dy*w with reinterpret_cast by casting to float4 for speedup
  // 2. Compute xhat with reinterpret_cast by casting to float4 for speedup
  // 3. Compute reduce sum for dxhat and dxhat*xhat with blockReduce
  // 4. Compute final gradient

  // step 1
  int offset = blockIdx.x * hidden_dim + threadIdx.x;
  float4 dxhat, xhat;
  float var_rsqrt;
  if (threadIdx.x < hidden_dim) {
    dxhat = ((const float4 *)out_grad)[offset];
    float4 vgamma = ((const float4 *)gamma)[threadIdx.x];
    dxhat.x *= vgamma.x;
    dxhat.y *= vgamma.y;
    dxhat.z *= vgamma.z;
    dxhat.w *= vgamma.w;

    // step 2
    xhat = ((const float4 *)inp)[offset];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);
    if (means == nullptr) {
      float4 vbetta = ((const float4 *)betta)[threadIdx.x];
      xhat.x = (xhat.x - vbetta.x) / add_eps(vgamma.x);
      xhat.y = (xhat.y - vbetta.y) / add_eps(vgamma.y);
      xhat.z = (xhat.z - vbetta.z) / add_eps(vgamma.z);
      xhat.w = (xhat.w - vbetta.w) / add_eps(vgamma.w);
    } 
    else {
      float fmean = (float)means[blockIdx.x];
      xhat.x = (xhat.x - fmean) * var_rsqrt;
      xhat.y = (xhat.y - fmean) * var_rsqrt;
      xhat.z = (xhat.z - fmean) * var_rsqrt;
      xhat.w = (xhat.w - fmean) * var_rsqrt;
    }
  }

  // step 3
  float reduce_val[2] = {0.f, 0.f};
  if (threadIdx.x < hidden_dim) {
    reduce_val[0] = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
    reduce_val[1] = dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z +
                    dxhat.w * xhat.w;
  }
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 4;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  // step 4
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  dxhat.x = (dxhat.x - s_sum_dxhat - xhat.x * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.y = (dxhat.y - s_sum_dxhat - xhat.y * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.z = (dxhat.z - s_sum_dxhat - xhat.z * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.w = (dxhat.w - s_sum_dxhat - xhat.w * s_sum_dxhat_xhat) * var_rsqrt;

  ((float4 *)inp_grad)[offset] = dxhat;
  
  /// END ASSIGN3_2
}
extern "C" {
void launch_layernorm_bw(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *inp, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch_size, int hidden_dim,
                         hipStream_t stream_1, hipStream_t stream_2) {
  
  // Allocate device memory
  float *d_gamma_grad, *d_betta_grad, *d_inp_grad, *d_out_grad, *d_inp, *d_gamma, *d_betta, *d_vars, *d_means;
  int grad_output_size = batch_size * hidden_dim * sizeof(float);
  int gamma_betta_size = hidden_dim * sizeof(float);
  int vars_means_size = batch_size * sizeof(float);

  hipMalloc((void **)&d_gamma_grad, gamma_betta_size);
  hipMalloc((void **)&d_betta_grad, gamma_betta_size);
  hipMalloc((void **)&d_inp_grad, grad_output_size);
  hipMalloc((void **)&d_out_grad, grad_output_size);
  hipMalloc((void **)&d_inp, grad_output_size);
  hipMalloc((void **)&d_gamma, gamma_betta_size);
  hipMalloc((void **)&d_betta, gamma_betta_size);
  hipMalloc((void **)&d_vars, vars_means_size);
  hipMalloc((void **)&d_means, vars_means_size);

  // Copy memory to device
  hipMemcpy((void *)d_out_grad, out_grad, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_inp, inp, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_gamma, gamma, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_betta, betta, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_vars, vars, vars_means_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_means, means, vars_means_size, hipMemcpyHostToDevice);

  // Launch kernels
  // Compute grad of gamma and betta
  // This calculates the number of blocks needed to cover the data along the specified dimension, rounds it up.
  // The result is then multiplied by TILE_DIM to ensure that the grid size is a multiple of TILE_DIM.
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream_1>>>(
      d_gamma_grad, d_betta_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars,
      d_means, batch_size, hidden_dim);

  // Compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch_size, nthread, 0, stream_2>>>(
      d_inp_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars, d_means, hidden_dim);

  // Synchronize and check for errors
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy back to host
  hipMemcpy(gamma_grad, d_gamma_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(betta_grad, d_betta_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(inp_grad, d_inp_grad, grad_output_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_gamma_grad);
  hipFree(d_betta_grad);
  hipFree(d_inp_grad);
  hipFree((void *)d_out_grad);
  hipFree((void *)d_inp);
  hipFree((void *)d_gamma);
  hipFree((void *)d_betta);
  hipFree((void *)d_vars);
  hipFree((void *)d_means);
}}
}}
